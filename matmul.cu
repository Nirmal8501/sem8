// #include <iostream>
// using namespace std;

// __global__ void multi(int *A, int *B, int *C, int n){
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;

//     if(row<n && col<n){
//         int sum = 0;
//         for(int k=0; k<n; k++){
//             sum += A[row*n + k] * B[k*n + col];
//         }
//         C[row*n + col] = sum;
//     }
// }

// int main(){
//     int N = 2;
//     int matrixSize = N*N;
//     size_t matrixBytes = matrixSize * sizeof(int);

//     int *a,*b,*c; //host matrix
//     a = new int[matrixSize];
//     b = new int[matrixSize];
//     c = new int[matrixSize];


//     int *x,*y,*z; // device matrix
//     cudaMalloc(&x, matrixBytes);
//     cudaMalloc(&x, matrixBytes);
//     cudaMalloc(&x, matrixBytes);

//     for(int i=0; i< N*N; i++){ //matrix initialise
//         a[i] = b[i] = i;
//     }

//     cudaMemcpy(x,a, matrixBytes, cudaMemcpyHostToDevice);
//     cudaMemcpy(y,b, matrixBytes, cudaMemcpyHostToDevice);
//     // cudaMemcpy(z,c, matrixBytes, cudaMemcpyHostToDevice);

//     int THREADS = 2;
//     int BLOCKS = N/THREADS;

//     dim3 threads(THREADS, THREADS);
//     dim3 blocks(BLOCKS, BLOCKS);

//     multi<<<blocks, threads >>> (x,y,z,N);

//     cudaMemcpy(a,x, matrixBytes, cudaMemcpyDeviceToHost);
//     cudaMemcpy(b,y, matrixBytes, cudaMemcpyDeviceToHost);
//     cudaMemcpy(c,z, matrixBytes, cudaMemcpyDeviceToHost);

//     for(int i=0; i<N*N; i++){
//         cout << a[i] << " ";
//     if ((i + 1) % N == 0) {
//         cout << "\n";
//     }

//     }

//     for(int i=0; i<N*N; i++){
//         cout << a[i] << " ";
//     if ((i + 1) % N == 0) {
//         cout << "\n";
//     }

//     }

//     for(int i=0; i<N*N; i++){
//         cout << a[i] << " ";
//     if ((i + 1) % N == 0) {
//         cout << "\n";
//     }

//     }

// }



#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void mul(int *a, int *b, int *c, int n){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if(row<n && col<n){
    int sum=0;
    for(int i=0; i<n; i++){
        sum += a[row * n + i] * b[i*n+col];
    }
    c[row*n+col]=sum;
  }

}

  int main(){

    int N= 3;

    int msize = N*N;
    size_t mbyte = msize*sizeof(int);

    int *a = new int[msize];
    int *b = new int[msize];
    int *c = new int[msize];

    int *x, *y, *z;
    hipMalloc(&x, mbyte);
    hipMalloc(&y, mbyte);
    hipMalloc(&z, mbyte);

    for(int i=0; i<msize; i++){
        a[i]=b[i]=i;
    }

    hipMemcpy(x,a, mbyte, hipMemcpyHostToDevice);
    hipMemcpy(y,b, mbyte, hipMemcpyHostToDevice);

    dim3 thread(16,16);
    dim3 block ((N + thread.x - 1)/thread.x, (N+thread.y-1)/thread.y);

    mul<<<block, thread>>>(x,y,z,N);

    hipMemcpy(c,z, mbyte, hipMemcpyDeviceToHost);

    cout<<endl;
    cout<<"Matrix A : "<<endl;
    for(int row=0; row<N; row++){
        for(int col=0; col<N; col++){
            cout<<a[row*N + col]<<" ";
        }
        cout<<endl;
    }

    cout<<endl;

    cout<<"Matrix B : "<<endl;
    for(int row=0; row<N; row++){
        for(int col=0; col<N; col++){
            cout<<b[row*N + col]<<" ";
        }
        cout<<endl;
    }

    cout<<endl;


    cout<<"Matrix C : "<<endl;
    for(int row=0; row<N; row++){
        for(int col=0; col<N; col++){
            cout<<c[row*N + col]<<" ";
        }
        cout<<endl;
    }

    cout<<endl;

    delete[] a;
    delete[] b;
    delete [] c;

    hipFree(x);
    hipFree(y);
    hipFree(z);
    return 0;
  
}